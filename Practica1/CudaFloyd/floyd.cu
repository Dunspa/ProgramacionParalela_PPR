#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <string.h>
#include <sys/time.h>
#include "Graph.h"

// CUDA runtime
//#include <hip/hip_runtime.h>

// helper functions and utilities to work with CUDA
//#include <helper_functions.h>
//#include <hip/hip_runtime_api.h>

using namespace std;

double cpuSecond() {
	struct timeval tp;
	gettimeofday(&tp, NULL);
	return((double)tp.tv_sec + (double)tp.tv_usec * 1e-6);
}

__global__ void floyd_kernel_1d(int * M, const int nverts, const int k) {
	int ij = threadIdx.x + blockDim.x * blockIdx.x;
    const int i = ij / nverts;
    const int j = ij - i * nverts;

    if (i < nverts && j < nverts) {
		int Mij = M[ij];

		if (i != j && i != k && j != k) { // Evitar los 0 de la matriz (evitar el mismo vertice)
			int Mikj = M[i * nverts + k] + M[k * nverts + j];
			Mij = (Mij > Mikj) ? Mikj : Mij;
			M[ij] = Mij;
		}
  	}
}

__global__ void floyd_kernel_2d(int * M, const int nverts, const int k) {
	int j = blockIdx.x * blockDim.x + threadIdx.x;	// Índice de filas de hebra
	int i = blockIdx.y * blockDim.y + threadIdx.y; 	// Índice de columnas de hebra

	int ij = i * nverts + j;	// Índice global del elemento en la matriz
	// Para poder acceder a los valores dentro de la matriz por fila y columna
	int i2 = ij / nverts;		// Fila correspondiente al índice global
	int j2 = ij - i2 * nverts; 	// Columna correspondiente al índice global

    if (i < nverts && j < nverts) {
		int Mij = M[i2 * nverts + j2];

		// Evitar los 0 de la matriz (evitar el mismo vertice)
		if (i != j && i != k && j != k) {
			int Mikj = M[i2 * nverts + k] + M[k * nverts + j2];
			Mij = (Mij > Mikj) ? Mikj : Mij;
			M[ij] = Mij;
		}
  	}
}

int main (int argc, char *argv[]) {
	if (argc != 3) {
		cerr << "Sintaxis: " << argv[0] << " <archivo de grafo>" << " <blocksize>" << endl;
		return(-1);
	}

	// Tamaño de bloque
	const int blocksize = atoi(argv[2]);
	
	// Obtener la información de la GPU
	//int devID;
	//hipDeviceProp_t props;
	hipError_t err;
	/*err = hipGetDevice(&devID);
  	if(err != hipSuccess) {
		cout << "ERROR AL OBTENER LA INFORMACIÓN DE LA GPU" << endl;
	}
	hipGetDeviceProperties(&props, devID);
	printf("Device %d: \"%s\" with Compute %d.%d capability\n", devID, props.name, props.major, props.minor);*/

	// Leer el grafo del archivo 
	Graph G;
	G.lee(argv[1]);

	//cout << "El Grafo de entrada es:" << endl;
	//G.imprime();

	const int nverts = G.getVertices();
	cout << nverts << " ";
	const int niters = nverts;
	const int nverts2 = nverts * nverts;

	int * c_Out_M = new int[nverts2];
	int size = nverts2*sizeof(int);
	int * d_In_M = NULL;

	err = hipMalloc((void **) &d_In_M, size);
	if (err != hipSuccess) {
		cout << "ERROR RESERVA" << endl;
	}

	int *A = G.Get_Matrix();

	// CPU phase
	double t1 = cpuSecond();

	// BUCLE PRINCIPAL DEL ALGORITMO
	int inj, in, kn;
	for (int k = 0 ; k < niters ; k++) {
        kn = k * nverts;

		for (int i = 0 ; i < nverts ; i++) {
			in = i * nverts;

			for (int j = 0; j < nverts; j++) {
				if (i!=j && i!=k && j!=k){
					inj = in + j;
					A[inj] = min(A[in+k] + A[kn+j], A[inj]);
				}
			}
		}
	}

  	double tcpu = cpuSecond() - t1;
	
	// Tiempo en CPU
  	cout << tcpu << " ";

	// GPU phase 1
	t1 = cpuSecond();

	err = hipMemcpy(d_In_M, A, size, hipMemcpyHostToDevice);
	if (err != hipSuccess) {
		cout << "ERROR COPIA A GPU" << endl;
	}

	for (int k = 0 ; k < niters ; k++) {
		// Tamaño del bloque (número de hebras por bloque)
	 	int threadsPerBlock = blocksize * blocksize;
		// Tamaño del grid (número de bloques por grid)
	 	int blocksPerGrid = (nverts2 + threadsPerBlock - 1) / threadsPerBlock;

	  	floyd_kernel_1d<<< blocksPerGrid, threadsPerBlock >>>(d_In_M, nverts, k);
	  	err = hipGetLastError();

	  	if (err != hipSuccess) {
	  		fprintf(stderr, "Failed to launch kernel 1! ERROR= %d\n",err);
	  		exit(EXIT_FAILURE);
		}
	}

	hipMemcpy(c_Out_M, d_In_M, size, hipMemcpyDeviceToHost);
	hipDeviceSynchronize();
	double tgpu1 = cpuSecond() - t1;

	// Tiempo en GPU con bloques unidimensionales
	cout << tgpu1 << " ";

	// GPU phase 2
	t1 = cpuSecond();

	err = hipMemcpy(d_In_M, A, size, hipMemcpyHostToDevice);
	if (err != hipSuccess) {
		cout << "ERROR COPIA A GPU" << endl;
	}

	for (int k = 0 ; k < niters ; k++) {
		// Tamaño del bloque (número de hebras por bloque)
		dim3 threadsPerBlock(blocksize, blocksize);
		// Tamaño del grid (número de bloques por grid)
		int tamx = ceil((float) (nverts) / threadsPerBlock.x);
		int tamy = ceil((float) (nverts) / threadsPerBlock.y);
		dim3 blocksPerGrid(tamx, tamy);

	  	floyd_kernel_2d<<< blocksPerGrid, threadsPerBlock >>>(d_In_M, nverts, k);
	  	err = hipGetLastError();

	  	if (err != hipSuccess) {
	  		fprintf(stderr, "Failed to launch kernel 2! ERROR= %d\n",err);
	  		exit(EXIT_FAILURE);
		}
	}

	hipMemcpy(c_Out_M, d_In_M, size, hipMemcpyDeviceToHost);
	hipDeviceSynchronize();
	double tgpu2 = cpuSecond() - t1;

	// Tiempo en GPU con bloques bidimensionales
	cout << tgpu2 << " ";

	// Ganancia en velocidad de ambas versiones GPU con respecto a la monohebra
	cout << tcpu / tgpu1 << " " << tcpu / tgpu2 << endl;

	for (int i = 0 ; i < nverts ; i++)
		for (int j = 0 ; j < nverts ; j++)
			if (abs(c_Out_M[i * nverts + j] - G.arista(i, j)) > 0)
				cout << "Error (" << i << "," << j << ")   " 
					<< c_Out_M[i * nverts + j] << "..." 
					<< G.arista(i,j) << endl;

	// Liberar toda la memoria
	free(c_Out_M);
	free(A);
	hipFree(d_In_M);
}
